#include "hip/hip_runtime.h"
/**
 *   Artur Romão e João Reis, May 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

# define N 1024


/* allusion to internal functions */

static double get_delta_time(void);

__global__ void mergeSubsequences(int* matrix_gpu, int subsequenceSize);

__global__ void sortSubsequence(int* matrix_gpu, int iter);

__device__ void swap(int* arr, int i, int j);

void validate(int *matrix);

void print_array(int arr[], int size, int file_size);

__global__ void print_device_array(int arr[], int size, int file_size);

bool contains(int *array, int size, int value);

void sort(int* matrix, int iter, int numSubsequences, int idx);

void swap_cpu(int* arr, int i, int j);

void merge(int* matrix, int iter, int numSubsequences, int idx);


int main (int argc, char **argv)  {

    printf("%s Starting...\n", argv[0]);
    if (sizeof (unsigned int) != (size_t) 4)
        return 1;                                             // it fails with prejudice if an integer does not have 4 bytes

    // set up the device
    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties (&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK (hipSetDevice (dev));

    // start counting the execution time
    (void) get_delta_time ();

    // create memory areas in host and device memory where the disk sectors data and sector numbers will be stored
    
    // host allocation memory
    int *matrix = (int *)malloc(N * N * sizeof(int));
    
    // device allocation memory
    int *device_matrix;
    CHECK (hipMalloc((void **) &device_matrix, sizeof(int) * N * N ));


    // initialize the host data
  
    // read the file
    const char * filename = "datSeq1M.bin";

    FILE *fp = fopen(filename, "r");
    
    if (fp == NULL) {
      printf("Error: could not open file %s\n", filename);
      return EXIT_FAILURE;
    }

    printf("ficheiro lido com sucesso!\n");

    // Read the header of the binary file
    int file_size;
    if (fread(&file_size, sizeof(int), 1, fp) != 1) {
        printf("Error reading the file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    printf("file_size =  %d\n", file_size);

    // Read the contents of the file
    int num;
    int i = 0;
    while(fread(&num, sizeof(int), 1, fp) == 1) {
        matrix[i] = num;
        i++;
    }

    fclose(fp);

    // copy the host data to the device memory
    (void) get_delta_time ();
    CHECK (hipMemcpy (device_matrix, matrix, N * N * sizeof(int), hipMemcpyHostToDevice));
    printf ("copy the host data to the device memory\n");
    
    // Sorting iterations
    int numSubsequences = N;

    dim3 grid(numSubsequences, 1);
    dim3 block(1, 1);


    for (int iteration = 0; iteration < 10; iteration++) {

        printf("\n>> iteration %d\n", iteration);

        if (iteration == 0) {
            sortSubsequence<<<grid, block>>>(device_matrix, iteration);

            // Wait for the sorting kernel to finish
            hipDeviceSynchronize();
        
            // print_device_array<<<1, 1>>>(device_matrix, N*N, N*N);
        } 

        numSubsequences /= 2;

        dim3 grid(numSubsequences, 1);
        dim3 block(1, 1);

        printf("starting merge\n");
        mergeSubsequences<<<grid, block>>>(device_matrix, iteration);

        // Wait for the merging kernel to finish
        hipDeviceSynchronize();
    }	

    int * sorted_matrix = (int *)malloc( sizeof(int) * N * N );
    CHECK (hipMemcpy (sorted_matrix, device_matrix, sizeof(int) * N * N, hipMemcpyDeviceToHost));

    validate(sorted_matrix);

    float exec_time = get_delta_time();
    printf("GPU execution time = %.6fs\n", exec_time);

    // free device global memory 
    CHECK (hipFree (device_matrix));

    // reset the device
    CHECK (hipDeviceReset ());

    // free host memory subseqmergeSize
    free(sorted_matrix);


    (void) get_delta_time ();
    numSubsequences = N;

    // CPU
    for (int iteration = 0; iteration < 10; iteration++) {

        printf("\n>> iteration %d\n", iteration);

        if (iteration == 0) {
            for (int idx = 0; idx < numSubsequences; ++idx) {
                sort(matrix, iteration, numSubsequences, idx);
            }
        } 

        numSubsequences /= 2;
        
        for (int idx = 0; idx < numSubsequences; ++idx) {
            merge(matrix, iteration, numSubsequences, idx);
        }
    }	

    validate(matrix);

    float exec_time_cpu = get_delta_time();
    printf("CPU execution time = %.6fs\n", exec_time_cpu);

    free(matrix);

    return 0;
}


__global__ void mergeSubsequences(int* matrix_gpu, int iter) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = blockDim.x * gridDim.x * y + x;  // Unique ID for each thread
    int subseq = 2 * N * (1 << iter) * idx;  // Starting index for each subsequence

    int subsequenceSize = (1 << iter) * N;  // Size of each subsequence
    int mergeSize = subsequenceSize * 2;  // Size of merged subsequences

    // Simple merge of two sorted subsequences
    int* subsequence1 = &matrix_gpu[subseq];
    int* subsequence2 = &matrix_gpu[subseq + subsequenceSize];
    
    int* temp = new int[mergeSize];

    // Merge operation
    int i = 0, j = 0, k = 0;
    while (i < subsequenceSize && j < subsequenceSize) {
        if (subsequence1[i] <= subsequence2[j]) {
            temp[k++] = subsequence1[i++];
        } else {
            temp[k++] = subsequence2[j++];
        }
    }
    // Copy the remaining elements of subsequence1[], if there are any
    while (i < subsequenceSize) {
        temp[k++] = subsequence1[i++];
    }
    // Copy the remaining elements of subsequence2[], if there are any
    while (j < subsequenceSize) {
        temp[k++] = subsequence2[j++];
    }

    __syncthreads();

    // Copy temp[] back to matrix_gpu
    for (i = 0; i < mergeSize; ++i) {
        matrix_gpu[subseq + i] = temp[i];
    }

    delete[] temp;  
}


__global__ void sortSubsequence(int* matrix_gpu, int iter) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = blockDim.x * gridDim.x * y + x;  // Unique ID for each thread
    int subseq = N * (1 << iter) * idx;  // Starting index for each subsequence

    // Perform sorting (bubble sort)
    for (int i = subseq; i < subseq + N; i++) {
        for (int j = i + 1; j < subseq + N; j++) {

            if (matrix_gpu[i] > matrix_gpu[j]) {
                swap(matrix_gpu, i, j);
            }
        }
    }

    __syncthreads();    
}

__device__ void swap(int* arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}


static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}


void validate(int *matrix) {

    int size = N * N;
    int i;
    for (i = 0; i < size; i++) {

        if (i == (size - 1))  {
            printf ("Everything is OK!\n");
            break;
        }

        if (matrix[i] > matrix[i+1]) { 
            printf ("Error in position %d between element %d and %d\n", i, matrix[i], matrix[i+1]);
            break;
        }
    }

    printf ("\n");
}


void print_array(int arr[], int size, int file_size) {
    for (int i = file_size - size; i < file_size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}


__global__ void print_device_array(int arr[], int size, int file_size) {
    for (int i = file_size - size; i < file_size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}


bool contains(int *array, int size, int value) {
    for (int i = 0; i < size; ++i) {
        if (array[i] == value) {
            return true;
        }
    }
    return false;
}



void sort(int* matrix, int iter, int numSubsequences, int idx) {
    int subseq = N * (1 << iter) * idx;  // Starting index for each subsequence

    // Perform sorting (example: bubble sort)
    for (int i = subseq; i < subseq + N; i++) {
        for (int j = i + 1; j < subseq + N; j++) {

            if (matrix[i] > matrix[j]) {
                swap_cpu(matrix, i, j);
            }
        }
    }
}

void merge(int* matrix, int iter, int numSubsequences, int idx) {
    int subseq = 2 * N * (1 << iter) * idx;  // Starting index for each subsequence

    int subsequenceSize = (1 << iter) * N;  // Size of each subsequence
    int mergeSize = subsequenceSize * 2;  // Size of merged subsequences

    // Simple merge of two sorted subsequences
    int* subsequence1 = &matrix[subseq];
    int* subsequence2 = &matrix[subseq + subsequenceSize];
    
    int* temp = new int[mergeSize];

    // Merge operation
    int i = 0, j = 0, k = 0;
    while (i < subsequenceSize && j < subsequenceSize) {
        if (subsequence1[i] <= subsequence2[j]) {
            temp[k++] = subsequence1[i++];
        } else {
            temp[k++] = subsequence2[j++];
        }
    }
    // Copy the remaining elements of subsequence1[], if there are any
    while (i < subsequenceSize) {
        temp[k++] = subsequence1[i++];
    }
    // Copy the remaining elements of subsequence2[], if there are any
    while (j < subsequenceSize) {
        temp[k++] = subsequence2[j++];
    }

    // Copy temp[] back to matrix
    for (i = 0; i < mergeSize; ++i) {
        matrix[subseq + i] = temp[i];
    }

    delete[] temp; 
}


void swap_cpu(int* arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}

