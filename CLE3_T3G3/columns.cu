#include "hip/hip_runtime.h"
/**
 *   Artur Romão e João Reis, May 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

# define N 8


/* allusion to internal functions */

static double get_delta_time(void);

__global__ void mergeSubsequences(int* matrix_gpu, int subsequenceSize);

__global__ void sortSubsequence(int* matrix_gpu, int iter);

__device__ void swap(int* arr, int i, int j);

void validate(int *matrix);

void print_array(int arr[], int size, int file_size);
__global__ void print_device_array(int arr[], int size, int file_size);
bool contains(int *array, int size, int value);


int main (int argc, char **argv)  {

    printf("%s Starting...\n", argv[0]);
    if (sizeof (unsigned int) != (size_t) 4)
        return 1;                                             // it fails with prejudice if an integer does not have 4 bytes

    // set up the device
    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties (&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK (hipSetDevice (dev));

    // create memory areas in host and device memory where the disk sectors data and sector numbers will be stored
    
    // alocar memória para a sequencia na memoria do cpu
    // reversar espaço na memoria do gpu
    
    // host allocation memory
    int *matrix = (int *)malloc(N * N * sizeof(int));
    
    // device allocation memory
    int *device_matrix;
    CHECK (hipMalloc((void **) &device_matrix, sizeof(int) * N * N ));


    // initialize the host data
  
    // leitura do ficheiro
    const char * filename = "datSeq1M.bin";

    FILE *fp = fopen(filename, "r");
    
    if (fp == NULL) {
      printf("Error: could not open file %s\n", filename);
      return EXIT_FAILURE;
    }

    printf("ficheiro lido com sucesso!\n");

    // Read the header of the binary file
    int file_size;
    if (fread(&file_size, sizeof(int), 1, fp) != 1) {
        printf("Error reading the file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    printf("file_size =  %d\n", file_size);

    // Read the contents of the file
    /* int num;
    int i = 0;
    while(fread(&num, sizeof(int), 1, fp) == 1) {
        matrix[i] = num;
        i++;
    }*/

    int count = 0;

    file_size = 64;
    // Generate unique random numbers
    while (count < file_size) {
        int randomNum = rand() % 100; // Generate a random number between 0 and 1000

        // Check if the random number is already in the array
        if (!contains(matrix, count, randomNum)) {
            matrix[count] = randomNum;
            count++;
        }
    }

    fclose(fp); 

    print_array(matrix, N*N, N*N);

    // copy the host data to the device memory
    (void) get_delta_time ();
    CHECK (hipMemcpy (device_matrix, matrix, N * N * sizeof(int), hipMemcpyHostToDevice));
    printf ("dados copiados do cpu para o gpu\n");
    
    // Sorting iterations
    int numSubsequences = N;

    dim3 grid(numSubsequences, 1);
    dim3 block(1, 1);


    for (int iteration = 0; iteration < 3; iteration++) {

        printf("\n>> iteration %d\n", iteration);

        if (iteration == 0) {
            sortSubsequence<<<grid, block>>>(device_matrix, iteration);

            // Wait for the sorting kernel to finish
            hipDeviceSynchronize();
        
            // print_device_array<<<1, 1>>>(device_matrix, N*N, N*N);
        } 

        numSubsequences /= 2;

        dim3 grid(numSubsequences, 1);
        dim3 block(1, 1);

        printf("starting merge\n");
        mergeSubsequences<<<grid, block>>>(device_matrix, iteration); // sizeof(int) * mergeSize is the specification of resulting merge array size
        
        // printf("device array = ");
        print_device_array<<<1, 1>>>(device_matrix, N*N, N*N);

        // Wait for the merging kernel to finish
        hipDeviceSynchronize();
    }	


    int * sorted_matrix = (int *)malloc( sizeof(int) * N * N );
    CHECK (hipMemcpy (sorted_matrix, device_matrix, sizeof(int) * N * N, hipMemcpyDeviceToHost));

    print_array(sorted_matrix, file_size, file_size);
    validate(sorted_matrix);

    // free device global memory 
    CHECK (hipFree (device_matrix));

    // reset the device
    CHECK (hipDeviceReset ());

    // free host memory subseqmergeSize
    free(matrix);
    free(sorted_matrix);

    return 0;
}


__global__ void mergeSubsequences(int* matrix_gpu, int iter) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = blockDim.x * gridDim.x * y + x;  // Unique ID for each thread

    int subsequenceSize = (1 << iter) * N;  // Size of each subsequence
    int mergeSize = subsequenceSize * 2;  // Size of merged subsequences
    
    int* temp = new int[mergeSize];

    // Merge operation
    int i = 0, j = 0, k = 0;
    while (i < subsequenceSize && j < subsequenceSize) {

        int subseq1_idx = 2 * ((1 << iter) * idx) + (N * (i % N)) + (i / N);
        int subseq2_idx = 2 * ((1 << iter) * idx) + 1 + (N * (j % N)) + (j / N);

        // if(iter == 1) printf("subseq1_idx = %d | subseq2_idx = %d\n", subseq1_idx, subseq2_idx);

        // printf("thread(%d) : matrix_gpu[%d] = %d  | matrix_gpu[%d] = %d  \n", idx, subseq1_idx, matrix_gpu[subseq1_idx], subseq2_idx, matrix_gpu[subseq2_idx]);

        if (matrix_gpu[subseq1_idx] <= matrix_gpu[subseq2_idx]) {
            temp[k++] = matrix_gpu[subseq1_idx];
            i++; 
        } else {
            temp[k++] = matrix_gpu[subseq2_idx];
            j++;
        }
    }

    while (i < subsequenceSize) {
        int subseq1_idx = 2 * ((1 << iter) * idx) + (N * (i % N)) + (i / N);
        //printf("[1thread %d](%d) - %d\n", idx, subseq1_idx, matrix_gpu[subseq1_idx]);
        temp[k++] = matrix_gpu[subseq1_idx];
        i++;
    }

    while (j < subsequenceSize) {
        int subseq2_idx =  2 * ((1 << iter) * idx) + 1 + (N * (j % N)) + (j / N);
        // printf("[2thread %d](%d) - %d\n", idx, subseq2_idx, matrix_gpu[subseq2_idx]);
        temp[k++] = matrix_gpu[subseq2_idx];
        j++;
    }

    __syncthreads();

    //int numCols = mergeSize / N;

    /* for (i = 0; i < mergeSize; i++) {
        int col = i / stepSize;
        int subseq_idx = ((1 << iter) * idx) + (N * col);
        subseq_idx = ((1 << iter) * idx) + (N * col);
        if (idx == 0) printf("subseq_idx = %d | temp[i] = %d\n", subseq_idx, temp[i]);
        matrix_gpu[subseq_idx] = temp[i];
    } */

    /* for (i = 0; i < mergeSize; i++) {
        int subseq_idx = (idx * mergeSize) + i * N;
        if (idx == 0) printf("subseq_idx = %d | temp[i] = %d\n", subseq_idx, temp[i]);
        matrix_gpu[subseq_idx] = temp[i];
    }
 */

    for (i = 0; i < mergeSize; i++) {
        int subseq_idx = i * (1 << (2 - iter)) + idx;
        if (idx == 0) printf("subseq_idx = %d | temp[i] = %d\n", subseq_idx, temp[i]);
        matrix_gpu[subseq_idx] = temp[i];
    }

    if (idx == 0) {
        printf("thread %d = ", idx);
        for (int i = 0; i < mergeSize; i++) printf("%d ", temp[i]);
        printf("\n");
    }

    delete[] temp; 
}



__global__ void sortSubsequence(int* matrix_gpu, int iter) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = blockDim.x * gridDim.x * y + x;  // Unique ID for each thread

    // Perform sorting (example: bubble sort)
    for (int i = 0; i < N; i++) {
        for (int j = i + 1; j < N; j++) {

            int subseq_idx = ((1 << iter) * idx) + (N * (i % N)) + (i / N);
            int next_subseq_idx = ((1 << iter) * idx) + (N * (j % N)) + (j / N);

            // printf("subseq_idx = %d | next_subseq_idx = %d\n", subseq_idx, next_subseq_idx);

            if (matrix_gpu[subseq_idx] > matrix_gpu[next_subseq_idx]) {
                swap(matrix_gpu, subseq_idx, next_subseq_idx);
            }
        }
    }

    __syncthreads();    
}

__device__ void swap(int* arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}


static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}


void validate(int *matrix) {

    int size = N * N;
    int i;
    for (i = 0; i < size; i++) {

        if (i == (size - 1))  {
            printf ("Everything is OK!\n");
            break;
        }

        if (matrix[i] > matrix[i+1]) { 
            printf ("Error in position %d between element %d and %d\n", i, matrix[i], matrix[i+1]);
            break;
        }
    }

    printf ("\n");
}


void print_array(int arr[], int size, int file_size) {
    for (int i = file_size - size; i < file_size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}


__global__ void print_device_array(int arr[], int size, int file_size) {
    for (int i = file_size - size; i < file_size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}


bool contains(int *array, int size, int value) {
    for (int i = 0; i < size; ++i) {
        if (array[i] == value) {
            return true;
        }
    }
    return false;
}